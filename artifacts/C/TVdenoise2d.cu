#include "hip/hip_runtime.h"
#define CUDA
#include "cuheader.h"
#define SQUARE(x) (_t=(x), _t*_t)
#define SMALL 1e-5f

__device__ void TVdenoise2d_fast_dev(USIZE_t i, USIZE_t j, float y[], const float x[], const USIZE_t d[], const float lambda[])
{
    USIZE_t ij = i + j*d[0], n = d[0]*d[1], k;
    float _t, w22, w12 = 0.0f, w32 = 0.0f, w23 = 0.0f;
    float *yp, yb[12][4], *ybp;
    SSIZE_t d0 = (SSIZE_t)d[0];

    for(k=0, yp=y+ij; k<d[2]; k++, yp+=n)
    {
        float lambda_k = lambda[k];
        float y12, y13, y21, y22, y23, y31, y32;

        /* Doesn't handle edges, so need i>=1 & i<d[0]-1 & j>=1 & j<d[1]-1 */
        /* Use one of the four different arrangements for how the centre voxel (y22)
           could be influenced by its neighbours. Fast version just uses the
           first.
           o   o   o  :   o    y12   y13
                      :
               |   |  :         |     |
           o --* --o  :  y21 --y22 --y23
                      :
               |      :         |
           o --o   o  :  y31 --y32    o
        */
                           y12   = yp[-1]; y13   = yp[-1+d0];
        y21   = yp[  -d0]; y22   = *yp   ; y23   = yp[  +d0];
        y31   = yp[ 1-d0]; y32   = yp[ 1];

        ybp    = yb[k];
        ybp[0] = y12;
        ybp[1] = y32;
        ybp[2] = y21;
        ybp[3] = y23;

        w12  += lambda_k*(SQUARE(y22-y21) + SQUARE(y22-y12) + SMALL); /* links to 21 and 12 */
        w23  += lambda_k*(SQUARE(y22-y23) + SQUARE(y13-y23) + SMALL); /* link to 23 */
        w32  += lambda_k*(SQUARE(y22-y32) + SQUARE(y31-y32) + SMALL); /* link to 32 */
    }

    /* See https://francisbach.com/the-%ce%b7-trick-or-the-effectiveness-of-reweighted-least-squares */
    /* w.. = 1/eta.. */
    w12 = 1.0f/sqrt(w12);
    w32 = 1.0f/sqrt(w32);
    w23 = 1.0f/sqrt(w23);
    w22 = w12 + w12 + w32 + w23;
/*
    for(k=0, yp=y; k<d[2]; k++, yp+=n, x+=n)
        yp[ij] = (lambda[k]*((yp[ij-1] + yp[ij-0])*w12 + yp[ij+1]*w32 + yp[ij+d0]*w23) + x[ij])
                /(lambda[k]*w22+1.0f);
*/
    x += ij;
    for(k=0, yp=y+ij; k<d[2]; k++, x+=n, yp+=n)
    {
        ybp = yb[k];
        *yp = (lambda[k]*((ybp[0] + ybp[2])*w12 + ybp[1]*w32 + ybp[3]*w23) + *x)
             /(lambda[k]*w22+1.0f);
    }
}


__device__ void TVdenoise2d_dev(USIZE_t i, USIZE_t j, float y[], const float x[], const USIZE_t d[], const float lambda[])
{
    USIZE_t ij = i + j*d[0], n = d[0]*d[1], k;
    float _t, w22, w12, w32, w21, w23;
    float eta[12];
    float *yp, yb[12][4], *ybp;
    SSIZE_t d0 = (SSIZE_t)d[0];

    for(k=0; k<12; k++) eta[k] = 0.0f;

    for(k=0, yp=y+ij; k<d[2]; k++, yp+=n)
    {
        float lambda_k = lambda[k];
        float y11, y12, y13, y21, y22, y23, y31, y32, y33;
        float d12, d21, d32, d23;

        /* Doesn't handle edges, so need i>=1 & i<d[0]-1 & j>=1 & j<d[1]-1 */
        y11   = yp[-1-d0]; y12   = yp[-1]; y13   = yp[-1+d0];
        y21   = yp[  -d0]; y22   =*yp    ; y23   = yp[   d0];
        y31   = yp[ 1-d0]; y32   = yp[ 1]; y33   = yp[ 1+d0];

        /* Save for later */
        ybp    = yb[k];
        ybp[0] = y12;
        ybp[1] = y32;
        ybp[2] = y21;
        ybp[3] = y23;

        /* Four different arrangements for how the centre voxel (y22) 
           could be influenced by its neighbours.
         o   o   o  :  o   o-- o  :  o --o   o  :  o   o   o
                    :      |      :      |      :
             |   |  :             :             :  |   |
         o --* --o  :  o-- *-- o  :  o --* --o  :  o-- *-- o
                    :  |   |      :      |   |  :
             |      :             :             :      |
         o --o   o  :  o   o   o  :  o   o   o  :  o   o-- o
        */

        d12 = SQUARE(y22-y12);
        d21 = SQUARE(y22-y21);
        d23 = SQUARE(y22-y23);
        d32 = SQUARE(y22-y32);

        eta[0]  += lambda_k*(d21 + d12             + SMALL);
        eta[1]  += lambda_k*(d23 + SQUARE(y13-y23) + SMALL);
        eta[2]  += lambda_k*(d32 + SQUARE(y31-y32) + SMALL);

        eta[3]  += lambda_k*(d23 + d32             + SMALL);
        eta[4]  += lambda_k*(d21 + SQUARE(y31-y21) + SMALL);
        eta[5]  += lambda_k*(d12 + SQUARE(y13-y12) + SMALL);

        eta[6]  += lambda_k*(d32 + d21             + SMALL);
        eta[7]  += lambda_k*(d23 + SQUARE(y33-y23) + SMALL);
        eta[8]  += lambda_k*(d12 + SQUARE(y11-y12) + SMALL);

        eta[9]  += lambda_k*(d23 + d12             + SMALL);
        eta[10] += lambda_k*(d21 + SQUARE(y11-y21) + SMALL);
        eta[11] += lambda_k*(d32 + SQUARE(y33-y32) + SMALL);

    }

    /* See https://francisbach.com/the-%ce%b7-trick-or-the-effectiveness-of-reweighted-least-squares */
    for(k=0; k<12; k++) eta[k] = 1.0f/sqrt(eta[k]);

    w12 = (eta[0] + eta[5] + eta[8] + eta[ 9])*0.25f;
    w21 = (eta[0] + eta[4] + eta[6] + eta[10])*0.25f;
    w32 = (eta[2] + eta[3] + eta[6] + eta[11])*0.25f;
    w23 = (eta[1] + eta[3] + eta[7] + eta[ 9])*0.25f;

    w22 = w12 + w21 + w32 + w23;

    x  += ij;
    for(k=0, yp=y+ij; k<d[2]; k++, x+=n, yp+=n)
    {
        ybp = yb[k];
        *yp = (lambda[k]*(ybp[0]*w12 + ybp[1]*w32 + ybp[2]*w21 + ybp[3]*w23) + *x)
             /(lambda[k]*w22+1.0f);
    }
}



/* Use constant memory for lower access times */
__constant__ USIZE_t o[2];    /* Offsets (x & y) */
__constant__ USIZE_t d[3];    /* Image dimensions (x & y) and number of images*/
__constant__ float lambda[8]; /* Regularisation of each channel */

__global__ void TVdenoise2d(float *y, const float *x)
{
    USIZE_t i, j;

    /* Leaves edge voxels alone */
    j = (blockIdx.y*blockDim.y + threadIdx.y)*3 + 1 + o[1];
    if (j>=d[1]-1) return;

    i = (blockIdx.x*blockDim.x + threadIdx.x)*3 + 1 + o[0];
    if (i>=d[0]-1) return;

    TVdenoise2d_dev(i, j, y, x, d, lambda);
}

__global__ void TVdenoise2d_fast(float *y, const float *x)
{
    USIZE_t i, j;

    /* Leaves edge voxels alone */
    j = (blockIdx.y*blockDim.y + threadIdx.y)*3 + 1 + o[1];
    if (j>=d[1]-1) return;

    i = (blockIdx.x*blockDim.x + threadIdx.x)*3 + 1 + o[0];
    if (i>=d[0]-1) return;

    TVdenoise2d_fast_dev(i, j, y, x, d, lambda);
}

