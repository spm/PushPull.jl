#include "hip/hip_runtime.h"
#define CUDA
#include "cuheader.h"
#define MAXVOL 20
#include "TVdenoise3d_dev.cu"

/* Use constant memory for lower access times */
__constant__ USIZE_t o[3];    /* Offsets (x & y) */
__constant__ float vox[3];    /* Voxel sizes */
__constant__ USIZE_t d[4];    /* Image dimensions (x & y) and number of images*/
__constant__ float lambdap[MAXVOL]; /* Regularisation of each channel */
__constant__ float lambdal[MAXVOL]; /* Reciprocals of variances of each channel */

__global__ void TVdenoise3d(float *y, const float *x)
{
    USIZE_t i, j, k;

    /* Leaves edge voxels alone */
    k = (blockIdx.z*blockDim.z + threadIdx.z)*3 + 1 + o[2];
    if (k>=d[2]-1) return;

    j = (blockIdx.y*blockDim.y + threadIdx.y)*3 + 1 + o[1];
    if (j>=d[1]-1) return;

    i = (blockIdx.x*blockDim.x + threadIdx.x)*3 + 1 + o[0];
    if (i>=d[0]-1) return;

    TVdenoise3d_dev(i, j, k, y, x, d, vox, lambdap, lambdal);
}

__global__ void TVdenoise3d_fast(float *y, const float *x)
{
    USIZE_t i, j, k;

    /* Leaves edge voxels alone */
    k = (blockIdx.z*blockDim.z + threadIdx.z)*3 + 1 + o[2];
    if (k>=d[2]-1) return;

    j = (blockIdx.y*blockDim.y + threadIdx.y)*3 + 1 + o[1];
    if (j>=d[1]-1) return;

    i = (blockIdx.x*blockDim.x + threadIdx.x)*3 + 1 + o[0];
    if (i>=d[0]-1) return;

    TVdenoise3d_fast_dev(i, j, k, y, x, d, vox, lambdap, lambdal);
}

