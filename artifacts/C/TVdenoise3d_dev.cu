#include "hip/hip_runtime.h"
#define SQUARE(x) (_t=(x), _t*_t)
#define SMALL 1e-8f 

__device__ void TVdenoise3d_fast_dev(USIZE_t i, USIZE_t j, USIZE_t k, float y[], const float x[], const USIZE_t d[],
                                     const float vox[3], const float lambdap[], const float lambdal[])
{
    SSIZE_t d0 = (SSIZE_t)d[0], d1 = (SSIZE_t)d[1], d01 = d0*d1;
    USIZE_t ijk = i + d0*(j + d1*k), n = d0*d1*d[2], m;
    float   _t, ws, w0 = 0.0f, w1 = 0.0f, w2 = 0.0f, w3 = 0.0f;
    float   *yp, yb[MAXVOL][6], *ybp;
    float rv0 = 1.0f/vox[0]/vox[0], rv1 = 1.0f/vox[1]/vox[1], rv2 = 1.0f/vox[2]/vox[2];

    for(m=0, yp=y+ijk; m<d[3]; m++, yp+=n)
    {
        float lambdap_m = lambdap[m];
        float yt[3][3][3];

        yt[0][0][0] = yp[-1-d0-d01]; yt[0][0][1] = yp[  -d0-d01]; yt[0][0][2] = yp[ 1-d0-d01];
        yt[0][1][0] = yp[-1   -d01]; yt[0][1][1] = yp[     -d01]; yt[0][1][2] = yp[ 1   -d01];
        yt[0][2][0] = yp[-1+d0-d01]; yt[0][2][1] = yp[  +d0-d01]; yt[0][2][2] = yp[ 1+d0-d01];

        yt[1][0][0] = yp[-1-d0    ]; yt[1][0][1] = yp[  -d0    ]; yt[1][0][2] = yp[ 1-d0    ];
        yt[1][1][0] = yp[-1       ]; yt[1][1][1] = *yp          ; yt[1][1][2] = yp[ 1       ];
        yt[1][2][0] = yp[-1+d0    ]; yt[1][2][1] = yp[   d0    ]; yt[1][2][2] = yp[ 1+d0    ];

        yt[2][0][0] = yp[-1-d0+d01]; yt[2][0][1] = yp[  -d0+d01]; yt[2][0][2] = yp[ 1-d0+d01];
        yt[2][1][0] = yp[-1   +d01]; yt[2][1][1] = yp[      d01]; yt[2][1][2] = yp[ 1   +d01];
        yt[2][2][0] = yp[-1+d0+d01]; yt[2][2][1] = yp[  +d0+d01]; yt[2][2][2] = yp[ 1+d0+d01];

        ybp    = yb[m];
        ybp[0] = yt[0][1][1];
        ybp[1] = yt[1][0][1];
        ybp[2] = yt[1][1][0];
        ybp[3] = yt[1][1][2];
        ybp[4] = yt[1][2][1];
        ybp[5] = yt[2][1][1];

        w0   += lambdap_m*(SQUARE(yt[1][1][1]-yt[0][1][1])*rv2 + SQUARE(yt[1][1][1]-yt[1][0][1])*rv1 + SQUARE(yt[1][1][1]-yt[1][1][0])*rv0 + SMALL);
        w1   += lambdap_m*(SQUARE(yt[1][1][1]-yt[1][1][2])*rv0 + SQUARE(yt[1][1][2]-yt[1][0][2])*rv1 + SQUARE(yt[1][1][2]-yt[0][1][2])*rv2 + SMALL);
        w2   += lambdap_m*(SQUARE(yt[1][1][1]-yt[1][2][1])*rv1 + SQUARE(yt[1][2][1]-yt[1][2][0])*rv0 + SQUARE(yt[1][2][1]-yt[0][2][1])*rv2 + SMALL);
        w3   += lambdap_m*(SQUARE(yt[1][1][1]-yt[2][1][1])*rv2 + SQUARE(yt[2][1][1]-yt[2][0][1])*rv1 + SQUARE(yt[2][1][1]-yt[2][1][0])*rv0 + SMALL);
    }

    /* See https://francisbach.com/the-%ce%b7-trick-or-the-effectiveness-of-reweighted-least-squares */
    w0 = 1.0f/sqrt(w0);
    w1 =  rv0/sqrt(w1);
    w2 =  rv1/sqrt(w2);
    w3 =  rv2/sqrt(w3);
    ws = w0*(rv2 + rv1 + rv0) + w1 + w2 + w3;

    x += ijk;
    for(m=0, yp=y+ijk; m<d[3]; m++, x+=n, yp+=n)
    {
        ybp = yb[m];
        *yp = (lambdap[m]*((ybp[0]*rv2 + ybp[1]*rv1 + ybp[2]*rv0)*w0 + ybp[3]*w1 + ybp[4]*w2 + ybp[5]*w3) + *x*lambdal[m])
             /(lambdap[m]*ws + lambdal[m]);
    }
}




__device__ void TVdenoise3d_dev(USIZE_t i, USIZE_t j, USIZE_t k, float y[], const float x[], const USIZE_t d[],
                                const float vox[3], const float lambdap[], const float lambdal[])
{
    SSIZE_t d0 = (SSIZE_t)d[0], d1 = (SSIZE_t)d[1], d01 = d0*d1;
    USIZE_t ijk = i + d0*(j + d1*k), n = d0*d1*d[2], m;
    float   w111,  w011, w211, w101, w121, w110, w112;
    float   *yp, yb[MAXVOL][6], *ybp;
    float eta[32];
    float rv0 = 1.0f/vox[0]/vox[0], rv1 = 1.0f/vox[1]/vox[1], rv2 = 1.0f/vox[2]/vox[2];
    /*
        Solving the L1 regularisation problem involves minimising |y| = \frac{y^2}{2\eta} + \frac{\eta}{2}.
        This is achieved by alternating between:
            \hat{\eta} = |\hat{y}|
        and:
            \hat{y} = \argmin_y \frac{y^2}{2\hat{\eta}}

       Prblem here involves multi-channel TV, so instead of |y|, we could minimise
           \sqrt{\sum_m \lambdap_m ((y_{i,j,k,m}-y_{i+1,j,k,m})^2 + (y_{i,j,k,m}-y_{i,j+1,k,m})^2 + (y_{i,j,k,m}-y_{i,j,k+1,m})^2))}

       Also note that the above configuration could involve eight different permutations of neighbours.
       If the above neighbourhood configuration is denoted by +++, we also have ++-, +-+, +--, -++, -+-, ---, --+.
       Therefore, in practice this code uses the average of the eight possible neighbourhood configurations.
     */
    for(m=0; m<32; m++) eta[m] = 0.0f;

    for(m=0, yp=y+ijk; m<d[3]; m++, yp+=n)
    {
        float lambdap_m = lambdap[m];
        float yt[3][3][3];
        float yc, _t, t, s, t0a, t0b, t1a, t1b, t2a, t2b;

        /* Get the 3x3x3 patch */
        yt[0][0][0] = yp[-1-d0-d01]; yt[0][0][1] = yp[  -d0-d01]; yt[0][0][2] = yp[ 1-d0-d01];
        yt[0][1][0] = yp[-1   -d01]; yt[0][1][1] = yp[     -d01]; yt[0][1][2] = yp[ 1   -d01];
        yt[0][2][0] = yp[-1+d0-d01]; yt[0][2][1] = yp[  +d0-d01]; yt[0][2][2] = yp[ 1+d0-d01];

        yt[1][0][0] = yp[-1-d0    ]; yt[1][0][1] = yp[  -d0    ]; yt[1][0][2] = yp[ 1-d0    ];
        yt[1][1][0] = yp[-1       ]; yt[1][1][1] = *yp          ; yt[1][1][2] = yp[ 1       ];
        yt[1][2][0] = yp[-1+d0    ]; yt[1][2][1] = yp[   d0    ]; yt[1][2][2] = yp[ 1+d0    ];

        yt[2][0][0] = yp[-1-d0+d01]; yt[2][0][1] = yp[  -d0+d01]; yt[2][0][2] = yp[ 1-d0+d01];
        yt[2][1][0] = yp[-1   +d01]; yt[2][1][1] = yp[      d01]; yt[2][1][2] = yp[ 1   +d01];
        yt[2][2][0] = yp[-1+d0+d01]; yt[2][2][1] = yp[  +d0+d01]; yt[2][2][2] = yp[ 1+d0+d01];

        yc     = *yp;         /* Central voxel */
        ybp    = yb[m];       /* Pointer for this volume */
        ybp[0] = yt[0][1][1]; /* -z */
        ybp[1] = yt[2][1][1]; /* +z */
        ybp[2] = yt[1][0][1]; /* -y */
        ybp[3] = yt[1][2][1]; /* +y */
        ybp[4] = yt[1][1][0]; /* -x */
        ybp[5] = yt[1][1][2]; /* +x */

        /* Immediate six neighbours */
        t2a    = SQUARE(yc-ybp[0])*rv2;
        t2b    = SQUARE(yc-ybp[1])*rv2;
        t1a    = SQUARE(yc-ybp[2])*rv1;
        t1b    = SQUARE(yc-ybp[3])*rv1;
        t0a    = SQUARE(yc-ybp[4])*rv0;
        t0b    = SQUARE(yc-ybp[5])*rv0;

        /* Centre voxel connecting to three neighbours (eight permutations). */
        s      = t1a + t2a + SMALL; eta[0] += lambdap_m*(s + t0a); eta[1] += lambdap_m*(s + t0b);
        s      = t1b + t2a + SMALL; eta[2] += lambdap_m*(s + t0a); eta[3] += lambdap_m*(s + t0b);
        s      = t1a + t2b + SMALL; eta[4] += lambdap_m*(s + t0a); eta[5] += lambdap_m*(s + t0b);
        s      = t1b + t2b + SMALL; eta[6] += lambdap_m*(s + t0a); eta[7] += lambdap_m*(s + t0b);

        /* Centre voxel connecting to one neighbour, but that neighbour connects to two other voxels
           (four permutations for each connecting neighbour). */
        t        = ybp[0];  /* -z neighbour */
        s        = SMALL  + t2a + SQUARE(t-yt[0][0][1])*rv1;
        eta[8]  += lambdap_m*(s + SQUARE(t-yt[0][1][0])*rv0);
        eta[9]  += lambdap_m*(s + SQUARE(t-yt[0][1][2])*rv0);
        s        = SMALL  + t2a + SQUARE(t-yt[0][2][1])*rv1;
        eta[10] += lambdap_m*(s + SQUARE(t-yt[0][1][0])*rv0);
        eta[11] += lambdap_m*(s + SQUARE(t-yt[0][1][2])*rv0);

        t        = ybp[1];  /* +z */
        s        = SMALL  + t2b + SQUARE(t-yt[2][1][0])*rv1;
        eta[12] += lambdap_m*(s + SQUARE(t-yt[2][0][1])*rv0);
        eta[13] += lambdap_m*(s + SQUARE(t-yt[2][2][1])*rv0);
        s        = SMALL  + t2b + SQUARE(t-yt[2][1][2])*rv1;
        eta[14] += lambdap_m*(s + SQUARE(t-yt[2][0][1])*rv0);
        eta[15] += lambdap_m*(s + SQUARE(t-yt[2][2][1])*rv0);

        t        = ybp[2];  /* -y */
        s        = SMALL  + t1a + SQUARE(t-yt[0][0][1])*rv2;
        eta[16] += lambdap_m*(s + SQUARE(t-yt[1][0][0])*rv0);
        eta[17] += lambdap_m*(s + SQUARE(t-yt[1][0][2])*rv0);
        s        = SMALL  + t1a + SQUARE(t-yt[2][0][1])*rv2;
        eta[18] += lambdap_m*(s + SQUARE(t-yt[1][0][0])*rv0);
        eta[19] += lambdap_m*(s + SQUARE(t-yt[1][0][2])*rv0);

        t        = ybp[3];  /* +y */
        s        = SMALL  + t1b + SQUARE(t-yt[0][2][1])*rv2;
        eta[20] += lambdap_m*(s + SQUARE(t-yt[1][2][0])*rv0);
        eta[21] += lambdap_m*(s + SQUARE(t-yt[1][2][2])*rv0);
        s        = SMALL  + t1b + SQUARE(t-yt[2][2][1])*rv2;
        eta[22] += lambdap_m*(s + SQUARE(t-yt[1][2][0])*rv0);
        eta[23] += lambdap_m*(s + SQUARE(t-yt[1][2][2])*rv0);

        t        = ybp[4];  /* -x */
        s        = SMALL +  t0a + SQUARE(t-yt[0][1][0])*rv2;
        eta[24] += lambdap_m*(s + SQUARE(t-yt[1][0][0])*rv1);
        eta[25] += lambdap_m*(s + SQUARE(t-yt[1][2][0])*rv1);
        s        = SMALL +  t0a + SQUARE(t-yt[2][1][0])*rv2;
        eta[26] += lambdap_m*(s + SQUARE(t-yt[1][0][0])*rv1);
        eta[27] += lambdap_m*(s + SQUARE(t-yt[1][2][0])*rv1);

        t        = ybp[5];  /* +x */
        s        = SMALL +  t0b + SQUARE(t-yt[0][1][2])*rv2;
        eta[28] += lambdap_m*(s + SQUARE(t-yt[1][0][2])*rv1);
        eta[29] += lambdap_m*(s + SQUARE(t-yt[1][2][2])*rv1);
        s        = SMALL +  t0b + SQUARE(t-yt[2][1][2])*rv2;
        eta[30] += lambdap_m*(s + SQUARE(t-yt[1][0][2])*rv1);
        eta[31] += lambdap_m*(s + SQUARE(t-yt[1][2][2])*rv1);
    }

    /* See https://francisbach.com/the-%ce%b7-trick-or-the-effectiveness-of-reweighted-least-squares */
    for(m=0; m<32; m++) eta[m] = 1.0f/sqrt(eta[m]);

    /* Weights from an average of the eight different arrangements of neighbours. */
    w011 = rv2*(eta[ 8]+eta[ 9]+eta[10]+eta[11] + eta[0]+eta[1]+eta[2]+eta[3])*0.125f; /* -z neighbour */
    w211 = rv2*(eta[12]+eta[13]+eta[14]+eta[15] + eta[4]+eta[5]+eta[6]+eta[7])*0.125f; /* +z */
    w101 = rv1*(eta[16]+eta[17]+eta[18]+eta[19] + eta[0]+eta[1]+eta[4]+eta[5])*0.125f; /* -y */
    w121 = rv1*(eta[20]+eta[21]+eta[22]+eta[23] + eta[2]+eta[3]+eta[6]+eta[7])*0.125f; /* +y */
    w110 = rv0*(eta[24]+eta[25]+eta[26]+eta[27] + eta[0]+eta[2]+eta[4]+eta[6])*0.125f; /* -x */
    w112 = rv0*(eta[28]+eta[29]+eta[30]+eta[31] + eta[1]+eta[3]+eta[5]+eta[7])*0.125f; /* +x */
    w111 = w011 + w211 + w101 + w121 + w110 + w112; /* Centre weight */

    x += ijk;
    for(m=0, yp=y+ijk; m<d[3]; m++, x+=n, yp+=n)
    {
        ybp = yb[m];
        *yp = (lambdap[m]*(ybp[0]*w011 + ybp[1]*w211 + ybp[2]*w101 + ybp[3]*w121 + ybp[4]*w110 + ybp[5]*w112) + *x*lambdal[m])
             /(lambdap[m]*w111 + lambdal[m]);
    }
}

