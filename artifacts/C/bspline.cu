#include "hip/hip_runtime.h"
#include "cuheader.h"

/*
static float wt0(float x)
{
    return((fabsf(x) > 0.5f) ? 0.0f : 1.0f);
}

static float wt1(float x)
{
    x = fabsf(x);
    return((x > 1.0f) ? 0.0f : (1.0f - x));
}
*/

__device__ float wt2(float x)
{
    x = fabsf(x);
    if(x < 0.5f)
        return(0.75f - x*x);
    if(x < 1.5f)
    {
        x = 1.5f - x;
        return(0.5f*x*x);
    }
    return(0.0f);
}

__device__ float wt3(float x)
{
    x = fabsf(x);
    if(x < 1.0f)
        return(x*x*(x - 2.0f)*0.5f + 2.0f/3.0f);
    if(x < 2.0f)
    {
        x = 2.0f - x;
        return(x*x*x*(1.0f/6.0f));
    }
    return(0.0f);
}

__device__ float wt4(float x)
{
    x = fabsf(x);
    if(x < 0.5f)
    {
        x *= x;
        return(x*(x*0.25f - 0.625f) + 115.0f/192.0f);
    }
    if(x < 1.5f)
        return(x*(x*(x*(5.0f/6.0f - x*(1.0f/6.0f)) - 1.25f) + 5.0f/24.0f) + 55.0f/96.0f);
    if(x < 2.5f)
    {
        x -= 2.5f;
        x *= x;
        return(x*x*(1.0f/24.0f));
    }
    return(0.0);
}


/* Note that dp[i] is 1 more than the interpolation degree */
__device__ SSIZE_t weights(const USIZE_t d, float x, /*@OUT@*/float w[])
{
    USIZE_t k;
    SSIZE_t i = (SSIZE_t)ceilf(x-0.5f*(float)d);
    x -= (float)i;

    switch (d){
    case 2:
        w[0] = 1.0f-x;
        w[1] = x;
        break;
    case 1:
        w[0] = 1.0f;
        break;
    case 3:
        for(k=0; k<=2; k++) w[k] = wt2(x-(float)k);
        break;
    case 4:
        for(k=0; k<=3; k++) w[k] = wt3(x-(float)k);
        break;
    case 5:
        for(k=0; k<=4; k++) w[k] = wt4(x-(float)k);
        break;
    default:
        w[0] = 1.0f-x;
        w[1] = x;
    }
    return i;
}


__device__ float dwt2(float x)
{
    int s;
    s = (x>0 ? 1 : -1);
    x = fabsf(x);

    if(x < 0.5f)
        return(-2*x*s);
    if(x < 1.5f)
        return((x - 1.5f)*s);
    return(0.0f);
}

__device__ float dwt3(float x)
{
    int s;
    s = (x>0 ? 1 : -1);
    x = fabsf(x);

    if(x < 1.0f)
        return(x*(1.5f*x - 2.0f)*s);
    if(x < 2.0f)
    {
        x = x - 2.0f;
        return(-0.5f*x*x*s);
    }
    return(0.0f);
}

__device__ float dwt4(float x)
{
    int s;
    s = (x>0 ? 1 : -1);
    x = fabsf(x);

    if(x < 0.5f)
        return((x*(x*x - 5.0f/4.0f))*s);
    if(x < 1.5f)
        return((x*(x*(x*(-2.0f/3.0f) + 2.5f) - 5.0f/2.0f) + 5.0f/24.0f)*s);
    if(x < 2.5f)
    {
        x = x*2.0f - 5.0f;
        return((1.0f/48.0f)*x*x*x*s);
    }
    return(0.0f);
}


/* Note that dp[i] is 1 more than the interpolation degree */
__device__ SSIZE_t dweights(const USIZE_t d, float x, /*@OUT@*/float w[])
{
    USIZE_t k;
    SSIZE_t i = (SSIZE_t)ceilf(x-0.5f*(float)d);
    x -= (float)i;

    switch (d){
    case 2:
        w[0] = -1.0f;
        w[1] =  1.0f;
        break;
    case 1:
        w[0] = 0.0f;
        break;
    case 3:
        for(k=0; k<=2; k++) w[k] = dwt2(x-(float)k);
        break;
    case 4:
        for(k=0; k<=3; k++) w[k] = dwt3(x-(float)k);
        break;
    case 5:
        for(k=0; k<=4; k++) w[k] = dwt4(x-(float)k);
        break;
    default:
        w[0] = -1.0f;
        w[1] =  1.0f;
    }
    return i;
}

/*
__device__ float hwt2(float x)
{
    x = fabsf(x);

    if(x < 0.5f)
        return(-2.0f);
    if(x < 1.5f)
        return(1.0f);
    return(0.0f);
}
*/

__device__ float hwt3(float x)
{
    x = fabsf(x);

    if(x < 1.0f)
        return(3.0f*x - 2.0f);
    if(x < 2.0f)
        return(2.0f - x);
    return(0.0f);
}

__device__ float hwt4(float x)
{
    x = fabsf(x);

    if(x < 0.5f)
        return(3.0f*x*x - 5.0f/4.0f);
    if(x < 1.5f)
        return(x*(-2.0f*x + 5.0f) - 5.0f/2.0f);
    if(x < 2.5f)
    {
        x = x*2.0f - 5.0f;
        return(x*x/8.0f);
    }
    return(0.0f);
}

__device__ SSIZE_t hweights(const USIZE_t d, float x, /*@OUT@*/float w[])
{
    USIZE_t k;
    SSIZE_t i = (SSIZE_t)ceilf(x-0.5f*(float)d);
    x -= (float)i;

    switch (d){
    case 2:
        w[0] = 0.0f;
        w[1] = 0.0f;
        break;
    case 1:
        w[0] = 0.0f;
        break;
    case 3:
     /* for(k=0; k<=2; k++) w[k] = hwt2(x-(float)k); */
        w[0] =  1.0f;
        w[1] = -2.0f;
        w[2] =  1.0f;
        break;
    case 4:
        for(k=0; k<=3; k++) w[k] = hwt3(x-(float)k);
        break;
    case 5:
        for(k=0; k<=4; k++) w[k] = hwt4(x-(float)k);
        break;
    default:
        w[0] = 0.0f;
        w[1] = 0.0f;
    }
    return i;
}

