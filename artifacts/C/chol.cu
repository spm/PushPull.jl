#include "hip/hip_runtime.h"
/* Cholesky decomposition
 * n  - dimension of matrix a
 * a  - an n \times n matrix
 * p  - an n \times 1 vector
 *
 * A triangle of the input matrix is partially overwritten
 * by the output. Diagonal elements are stored in p.
 */
__device__ void choldcf(USIZE_t n, float a[], /*@out@*/ float p[])
{
    USIZE_t i, j;
    SSIZE_t k;

    for(i=0; i<n; i++)
    {
        float *ai = a + i*n;
        for(j=i; j<n; j++)
        {
            float  t  = ai[j];
            float *aj = a + j*n;
            for(k=(SSIZE_t)i-1; k>=0; k--)
               t -= ai[k]*aj[k];
            if(j==i)
                p[i] = sqrtf(t);
            else
                aj[i] = t/p[i];
        }
    }
}


/* Solve a least squares problem with the results from a
 * Cholesky decomposition
 *
 * n     - Dimension of matrix and data.
 * a & p - Cholesky decomposed matrix.
 * b     - Vector of input data.
 * x     - Vector or outputs.
 */
__device__ void chollsf(USIZE_t n, const float a[], const float p[],
                        const float b[], /*@out@*/ float x[])
{
    SSIZE_t i, k;

    for(i=0; i<(SSIZE_t)n; i++)
    {
        float t = b[i];
        for(k=i-1; k>=0; k--)
            t -= a[i*n+k]*x[k];
        x[i] = t/p[i];
    }
    for(i=(SSIZE_t)n-1; i>=0; i--)
    {
        float t = x[i];
        for(k=i+1; k<(SSIZE_t)n; k++)
            t -= a[k*n+i]*x[k];
        x[i] = t/p[i];
    }
}

