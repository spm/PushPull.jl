#include "hip/hip_runtime.h"
#define CUDA
#include "cuheader.h"
#include "lbessi_dev.cu"

__device__ USIZE_t calculateGlobalIndex() {
    USIZE_t const globalBlockIndex = blockIdx.x + blockIdx.y * gridDim.x;
    USIZE_t const localThreadIdx   = threadIdx.x + blockDim.x * threadIdx.y;
    USIZE_t const threadsPerBlock  = blockDim.x*blockDim.y;
    return localThreadIdx + globalBlockIndex*threadsPerBlock;
}

__global__ void lbessi_element(float *out, const float nu, const float *z, const USIZE_t numel)
{
    USIZE_t const i = calculateGlobalIndex();
    if (i >= numel) return;
    out[i] = lbessif(nu,z[i]);
}

