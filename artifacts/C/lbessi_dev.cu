#include "hip/hip_runtime.h"
#include "cuheader.h"
/* Code based on
   "On the use of the noncentral chi-square density function
   for the distribution of helicopter spectral estimates."
   Garber DP.
   NASA, Langley Research Center (1993).
*/

/* Compute log(bessi(nu, z))
   - requires z to be reasonably small to work efficiently
*/
__device__ float lbessif_small(float const nu, float const z)
{
    USIZE_t M = 1000000, m;
    float x, f = 0.0, s = 0.0, so, y, g;
    x  = logf(0.5f*z);
    g  = lgammaf(nu+1.0f); /* gammaln(1) + gammaln(1+nu) */
    s  = 1.0f;
    f  = x * nu - g;
    for(m=1; m<=M; m++)
    {
        g += logf(m*(m + nu));
        y  = x * (2*m + nu) - g;
        if (y>f)
        {
            s *= expf(f - y);
            s += 1.0f;
            f  = y;
        }
        else
        {
            so = s;
            s += expf(y - f);
            if (so==s) break;
        }
        /* gammaln((m+1) + 1) + gammaln((m+1) + nu + 1) =
           gammaln( m    + 1) + gammaln( m    + nu + 1) + log(m + 1) + log(m + nu + 1) =
           gammaln( m    + 1) + gammaln( m    + nu + 1) + log((m + 1)*(m + nu + 1)) */
    }
    return (float)(logf(s) + f);
}


/* Compute log(bessi(nu, z))
   - requires z to be reasonably large for accurate results.
   - fails for nu0 = 0.0 (or close to 0.0).
*/
__device__ float lbessif_large(float const nu, float const z)
{
    static float tiny = 1.4014e-45f;
    float f, t, tt, ttt, us, den, tmp;
    f  = z/nu;
    f *= f;
    if (f>4.0f)
    {
        tmp = sqrtf(1.0f+1.0f/f);
        t   = z*tmp/nu;
        f   = nu*(t - logf(nu/z+tmp));
    }
    else
    {
        tmp = sqrtf(1.0f+f);
        t   = (tmp>1.0f) ? tmp : 1.0f;
        f   = nu*(t + logf(z/(nu*(1.0f+tmp))));
    }

    t    = 1.0f/t;
    tt   = t*t;
    ttt  = t*tt;
    us   = 1.0f;
    den  = nu;
    us  += t*(0.125f - tt*0.2083333333333333f)/den;
    den *= nu;
    us  += tt*(0.0703125f + tt*(-0.4010416666666667f + tt*0.3342013888888889f))/den;
    den *= nu;
    us  += ttt*(0.0732421875f + tt*(-0.8912109375f + tt*(1.846462673611111f - tt*1.025812596450617f)))/den;
    den *= nu;
    us  += tt*tt*(0.112152099609375f + tt*(-2.3640869140625f + tt*(8.78912353515625f +
           tt*(-11.20700261622299f + tt*4.669584423426248f))))/den;
    den *= nu;
    us  += tt*ttt*(0.2271080017089844f + tt*(-7.368794359479632f + tt*(42.53499874638846f +
           tt*(-91.81824154324002f + tt*(84.63621767460074f - tt*28.21207255820025f)))))/den;
    den *= nu;
    us  += ttt*ttt*(0.5725014209747314f + tt*(-26.49143048695155f + tt*(218.1905117442116f +
           tt*(-699.5796273761326f + tt*(1059.990452528f + tt*(-765.2524681411817f +
           tt*212.5701300392171f))))))/den;

    f   += 0.5*(logf((t < tiny) ? tiny : t) - logf(nu)) - 0.918938533204673f; /* 0.5*log(2*pi) */
    f   += logf((us<tiny) ? tiny : us);
    return f;
}


__device__ float lbessif(float const nu0, float const z0)
{
    float z = fabsf(z0), nu = fabsf(nu0);
    if (nu >= 15.0f)
        return lbessif_large(nu,z);
    else
    {
        float thr = 3.3333333333333333 * sqrtf(225.0f - nu*nu);
        if (z<thr)
            return lbessif_small(nu, z);
        else
        {
            nu = (nu < 1e-7) ? 1e-7 : nu;
            return lbessif_large(nu,z);
        }
    }
}


