#include "hip/hip_runtime.h"
#define CUDA
#include "cuheader.h"
#include "operator_dev.cu"
/*
    MAXD - maximum filter size
    MAXN - maximum number of gradient fields
*/

/*
    Use constant memory for faster access times
    Limited by available constant memory on device:
        hipDeviceAttributeTotalConstantMemory
*/
__constant__ float   l[MAXD*MAXD*MAXD*MAXN*(MAXN+1)/2];  /* Filters */
__constant__ float   lc[MAXN*(MAXN+1)/2]; /* sum over filter elements */
__constant__ int     bnd[3*MAXN];         /* Boundary conditions */
__constant__ USIZE_t dp[5];               /* filter dimensions */
__constant__ USIZE_t  o[3];               /* offsets into volume */
__constant__ USIZE_t  d[5];               /* image data dimensions */
__constant__ USIZE_t  n[3];               /* number of elements */


__global__ void relax_element(float *v, const float *g, const float *h)
{
    USIZE_t ijk, i, j, k;
    ijk = threadIdx.x + blockDim.x*blockIdx.x;
    i   = (ijk % n[0])*dp[0] + o[0]; if(i>=d[0]) return;
    ijk =  ijk / n[0];
    j   = (ijk % n[1])*dp[1] + o[1]; if(j>=d[1]) return;
    k   = (ijk / n[1])*dp[2] + o[2]; if(k>=d[2]) return;

    relaxN(i, j, k, v, d, g, h, dp, l, bnd);
}


__global__ void conv_element(float *u, const float *v)
{
    USIZE_t i, j, k;
    i = threadIdx.x + blockDim.x*blockIdx.x; if(i>=d[0]) return;
    j = threadIdx.y + blockDim.y*blockIdx.y; if(j>=d[1]) return;
    k = threadIdx.z + blockDim.z*blockIdx.z; if(k>=d[2]) return;

    convN(i, j, k, u, v, d, dp, l, lc, bnd);
}

