#include "cuheader.h"
#include "patch.cu"
#include "chol.cu"

/* Flexibility comes at the expense of speed because L is often mostly zeros.
   Dealing with padding is also likely to slow things down a lot, especially
   as many of the loaded voxels will not be used. */

__device__ float odconv(const float *lp, const float *vp, USIZE_t cp)
{
    const float *lpend;
    float vo   = 0.0f, vc = vp[cp];
    for(lpend=lp+cp; lp<lpend;) vo += *(lp++) * (*(vp++) - vc);
    vp++; /* Omit the central voxel */
    lp++;
    for(lpend=lp+cp; lp<lpend;) vo += *(lp++) * (*(vp++) - vc);
    return vo;
}


__device__ void relax1(USIZE_t i, USIZE_t j, USIZE_t k, float *v, const USIZE_t *d, const float *g, const float *h, const USIZE_t *dp, const float *l, const int *bnd)
{
    float vp[MAXD*MAXD*MAXD], b, a;
    SSIZE_t o[3];
    USIZE_t m, cp, hcode = d[4];
    o[0]  = (SSIZE_t)i-(SSIZE_t)dp[0]/2;
    o[1]  = (SSIZE_t)j-(SSIZE_t)dp[1]/2;
    o[2]  = (SSIZE_t)k-(SSIZE_t)dp[2]/2;

    cp    = (dp[0]*dp[1]*dp[2])/2;
    m     = i+d[0]*(j+d[1]*k);
    get_patch(dp, vp, bnd, o, d, v);
    b     = g[m] - odconv(l, vp, cp);
    a     = l[cp];
    if(hcode)
    {
        b -= h[m]*vp[cp];
        a += h[m];
    }
    v[m] += b/a;
}


#define MAXN 8

__device__ void relaxN(USIZE_t i, USIZE_t j, USIZE_t k, float *v, const USIZE_t *d, const float *g, const float *h, const USIZE_t *dp, const float *l, const int *bnd)
{
    float vp[MAXN*(MAXN+2)]; /* Also needs to big enough for prod(dp(1:3)) */
    SSIZE_t o[3];
    USIZE_t m, nd, np, cp, ii, d3 = d[3], hcode = d[4], lcode = dp[4];
    float *A = vp, *p = vp + MAXN*MAXN, *x = vp+MAXN*(MAXN+1); /* re-use memory */
    float b[MAXN];

    o[0] = i-(SSIZE_t)dp[0]/2;
    o[1] = j-(SSIZE_t)dp[1]/2;
    o[2] = k-(SSIZE_t)dp[2]/2;
    m    = i+d[0]*(j+d[1]*k);
    g   += m;
    h   += m;
    /* Original i, j & k no-longer needed */

    np   = dp[0]*dp[1]*dp[2];
    cp   = np/2;
    nd   = d[0]*d[1]*d[2];

    /* re-use i and j from here onwards */
    for(i=0; i<d3; i++)
        b[i] = g[nd*i];

    for(i=0, ii=d3; i<d3; i++)
    {
        USIZE_t j, ii0=ii;
        get_patch(dp, vp, bnd+i*3, o, d, v+i*nd);

        b[i] -= odconv(l+i*np, vp, cp);
        if(lcode==2)
        {
            for(j=i+1; j<d3; j++, ii++)
                b[j] -= 2.0f*odconv(l+ii*np, vp, cp);
        }
        if(hcode)
        {
            float vc = vp[cp];
            b[i] -= h[i*nd]*vc;

            if(hcode==2)
            {
                for(j=i+1, ii=ii0; j<d3; j++, ii++)
                    b[j] -= 2.0f*h[ii*nd]*vc;
            }
        }
    }

    /* Construct "diagonal" of L+H */
    l += cp;

    for(i=0; i<d3; i++)
        A[i+d3*i] = l[i*np]*1.000001f;
    if(lcode==2)
    {
        for(i=0, ii=d3; i<d3; i++)
        {
            USIZE_t j;
            for(j=i+1; j<d3; j++, ii++)
                A[i + j*d3] = A[j + i*d3] = l[ii*np];
        }
    }
    else
    {
        for(i=0, ii=d3; i<d3; i++)
        {
            USIZE_t j;
            for(j=i+1; j<d3; j++, ii++)
                A[i + j*d3] = A[j + i*d3] = 0.0f;
        }
    }

    if(hcode)
    {
        for(i=0; i<d3; i++)
            A[i+d3*i] += h[nd*i]*1.000001f;

        if(hcode==2)
        {
            for(i=0, ii=d3; i<d3; i++)
            {
                USIZE_t j;
                for(j=i+1; j<d3; j++, ii++)
                    A[i + j*d3] = A[j + i*d3] += h[ii*nd];
            }
        }
    }

    /* Compute x = A\b via Cholesky decomposition */
    choldcf(d3, A, p);
    chollsf(d3, A, p, b, x);

    v += m; /* shift pointer */
    for(i=0; i<d3; i++) v[nd*i] += x[i];
}

__device__ void convN(USIZE_t i, USIZE_t j, USIZE_t k, float *u, const float *v, const USIZE_t *d, const USIZE_t *dp, const float *l, const float *lc, const int *bnd)
{
    float vp[MAXD*MAXD*MAXD];
    SSIZE_t o[3];
    USIZE_t m, nd, np, cp, ii, d3 = d[3], lcode = dp[4];
    float b[MAXN];

    o[0] = i-(SSIZE_t)dp[0]/2;
    o[1] = j-(SSIZE_t)dp[1]/2;
    o[2] = k-(SSIZE_t)dp[2]/2;
    m    = i+d[0]*(j+d[1]*k);
    v   += m;
    np   = dp[0]*dp[1]*dp[2];
    cp   = np/2;
    nd   = d[0]*d[1]*d[2];

    /* re-use i and j from here onwards */
    for(i=0, ii=d3; i<d3; i++)
    {
        USIZE_t j;
        float vc = vp[cp];
        get_patch(dp, vp, bnd+i*3, o, d, v+i*nd);

        b[i] = odconv(l+i*np, vp, cp) + vc*lc[i];
        if(lcode==2)
        {
            for(j=i+1; j<d3; j++, ii++)
                b[j] += 2.0f*odconv(l+ii*np, vp, cp) + vc*lc[ii];
        }
    }

    u += m; /* shift pointer */
    for(i=0; i<d3; i++) u[nd*i] = b[i];
}

