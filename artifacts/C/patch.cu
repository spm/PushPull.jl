#include "hip/hip_runtime.h"
#ifndef MAXD
#    define MAXD 5
#endif

__device__ void circ_b(SSIZE_t i, USIZE_t d, /*@out@*/USIZE_t *b, /*@out@*/int *s)
{
    *b = (USIZE_t)((i%(SSIZE_t)d) + d) % d;
    *s = 1;
}


__device__ void refl_b(SSIZE_t i, USIZE_t d, /*@out@*/USIZE_t *b, /*@out@*/int *s)
{
    USIZE_t d2 = d<<1;
    USIZE_t i1 = ((USIZE_t)(i%(SSIZE_t)d2) + d2) % d2;
    *b = (i1<d) ? i1 : d2-(i1+1);
    *s = 1;
}


__device__ void refl1_b(SSIZE_t i, USIZE_t d, /*@out@*/USIZE_t *b, /*@out@*/int *s)
{
    USIZE_t d2 = d<<1;
    USIZE_t i1 = ((USIZE_t)(i%(SSIZE_t)d2) + d2) % d2;
    if(i1<d)
    {
        *b = i1;
        *s = 1;
    }
    else
    {
        *b = d2-(i1+1);
        *s = -1;
    }
}


__device__ void bound(const int bnd, const SSIZE_t o, const USIZE_t dp, const USIZE_t d, /*@out@*/USIZE_t b[], /*@out@*/int s[])
{
    USIZE_t i;
    switch(bnd)
    {
    case 0:
        for(i=0; i<dp; i++)
            circ_b((SSIZE_t)i+o, d, b+i, s+i);
        break;
    case 1:
        for(i=0; i<dp; i++)
            refl_b((SSIZE_t)i+o, d, b+i, s+i);
        break;
    case 2:
        for(i=0; i<dp; i++)
            refl1_b((SSIZE_t)i+o, d, b+i, s+i);
        break;
    default:
        for(i=0; i<dp; i++)
            refl_b((SSIZE_t)i+o, d, b+i, s+i);
    }
}


__device__ void get_patch(const USIZE_t dp[], /*@out@*/float *fp, const int bnd[], SSIZE_t o[], const USIZE_t d0[], const float *f0)
{
    USIZE_t i, j, k;
    USIZE_t kb[MAXD], jb[MAXD], ib[MAXD];
    int    ks[MAXD], js[MAXD], is[MAXD];
    bound(bnd[0], o[0], dp[0], d0[0], ib, is);
    bound(bnd[1], o[1], dp[1], d0[1], jb, js);
    bound(bnd[2], o[2], dp[2], d0[2], kb, ks);

    for(k=0; k<dp[2]; k++)
    {
        USIZE_t tk  = d0[1]*kb[k];
        int    ksk = ks[k];
        for(j=0; j<dp[1]; j++)
        {
            USIZE_t tj = d0[0]*(jb[j] + tk);
            int    s  = ksk*js[j];
            for(i=0; i<dp[0]; i++, fp++)
                *fp = f0[ib[i] + tj]*(int)(s*is[i]);
        }
    }
}

__device__ float diffs(/*@out@*/float g[3], const int bnd[], SSIZE_t o[], const USIZE_t d[], const float *f0)
{
    USIZE_t b[3], t1, t2;
    int     s[3];
    float f;

    bound(bnd[0], o[0]-1, 3, d[0], b, s);
    t1   =        d[0]*(o[1] + d[1]*o[2]);
    g[0] = (f0[b[2]    + t1]*s[2] - f0[b[0] + t1]*s[0]);
    f    =  f0[o[0]    + t1];

    bound(bnd[1], o[1]-1, 3, d[1], b, s);
    t1   = o[0] + d[0]*(       d[1]*o[2]);
    t2   = d[0];
    g[1] = (f0[b[2]*t2 + t1]*s[2] - f0[b[0]*t2 + t1]*s[0]);

    bound(bnd[2], o[2]-1, 3, d[2], b, s);
    t1   = o[0] + d[0]* o[1];
    t2  *= d[1];
    g[2] = (f0[b[2]*t2 + t1]*s[2] - f0[b[0]*t2 + t1]*s[0]);

    return f;
}


__device__ void put_patch(const USIZE_t dp[], const float *fp, const int bnd[], const SSIZE_t o[], const USIZE_t d0[], float *f0)
{
    USIZE_t i, j, k;
    USIZE_t kb[MAXD], jb[MAXD], ib[MAXD];
    int    ks[MAXD], js[MAXD], is[MAXD];
    bound(bnd[0], o[0], dp[0], d0[0], ib, is);
    bound(bnd[1], o[1], dp[1], d0[1], jb, js);
    bound(bnd[2], o[2], dp[2], d0[2], kb, ks);

    for(k=0; k<dp[2]; k++)
    {
        USIZE_t tk  = d0[1]*kb[k];
        int    ksk = ks[k];
        for(j=0; j<dp[1]; j++)
        {
            USIZE_t tj = d0[0]*(jb[j] + tk);
            int    s  = ksk*js[j];
            for(i=0; i<dp[0]; i++, fp++)
            {
                float t = *fp*(int)(s*is[i]);
/*
#ifdef C
#               pragma omp atomic
#endif
*/
                atomicAdd(&(f0[ib[i] + tj]),t);
            }
        }
    }
}

