#include "hip/hip_runtime.h"
#define CUDA
#include "cuheader.h"
#include "pushpull_dev.cu"
#include<hip/hip_math_constants.h>
#define ISFINITE(x) isfinite(x)

__device__ USIZE_t calculateGlobalIndex() {
    USIZE_t const globalBlockIndex = blockIdx.x + blockIdx.y * gridDim.x;
    USIZE_t const localThreadIdx   = threadIdx.x + blockDim.x * threadIdx.y;
    USIZE_t const threadsPerBlock  = blockDim.x*blockDim.y;
    return localThreadIdx + globalBlockIndex*threadsPerBlock;
}


/* Use constant memory for lower access times */
__constant__ int     bnd[3];  /* boundary codes   */
__constant__ USIZE_t  dp[3];  /* patch dimensions */
__constant__ USIZE_t  d0[3];  /* image dimensions */
__constant__ USIZE_t  n1;     /* Number of voxels */
__constant__ int     ext;     /* Extrapolate flag */

/* These are used by affine push/pull */
__constant__ USIZE_t  d1[3];  /* image dimensions */
__constant__ float   Aff[12]; /* Part of affine transform */


#define VOXOK(x, d0) (ISFINITE(x[0]) && ISFINITE(x[1]) && ISFINITE(x[2]) && \
                      (ext!=0 || (x[0]>=-0.01f && x[0]<=(float)(d0[0])-0.99f && \
                                  x[1]>=-0.01f && x[1]<=(float)(d0[1])-0.99f && \
                                  x[2]>=-0.01f && x[2]<=(float)(d0[2])-0.99f)))


__global__ void pull_element(float *f1, const float *phi, const float *f0)
{
    USIZE_t const globalThreadIdx = calculateGlobalIndex();
    float x[3];

    if(globalThreadIdx >= n1)
        return;

    x[0] = phi[globalThreadIdx]      - 1.0f;
    x[1] = phi[globalThreadIdx+n1]   - 1.0f;
    x[2] = phi[globalThreadIdx+n1*2] - 1.0f;

    if(VOXOK(x,d0))
        f1[globalThreadIdx] = pull1(d0, f0, bnd, dp, x);
    else
        f1[globalThreadIdx] = HIP_NAN_F;
}


__global__ void pullg_element(float *g1, const float *phi, const float *f0)
{
    USIZE_t const globalThreadIdx = calculateGlobalIndex();
    float x[3];

    if(globalThreadIdx >= n1) return;

    x[0] = phi[globalThreadIdx]      - 1.0f;
    x[1] = phi[globalThreadIdx+n1]   - 1.0f;
    x[2] = phi[globalThreadIdx+n1*2] - 1.0f;

    if(VOXOK(x,d0))
    {
        float g[3];
        pullg1(d0, f0, bnd, dp, x, g);
        g1[globalThreadIdx       ] = g[0];
        g1[globalThreadIdx + n1  ] = g[1];
        g1[globalThreadIdx + n1*2] = g[2];
    }
    else
    {
        g1[globalThreadIdx       ] = HIP_NAN_F;
        g1[globalThreadIdx + n1  ] = HIP_NAN_F;
        g1[globalThreadIdx + n1*2] = HIP_NAN_F;
    }
}


__global__ void pullh_element(float *h1, const float *phi, const float *f0)
{
    USIZE_t const globalThreadIdx = calculateGlobalIndex();
    float x[3];

    if(globalThreadIdx >= n1) return;

    x[0] = phi[globalThreadIdx]       - 1.0f;
    x[1] = phi[globalThreadIdx+n1]    - 1.0f;
    x[2] = phi[globalThreadIdx+n1*2]  - 1.0f;

    if(VOXOK(x,d0))
    {
        float h[6];
        pullh1(d0, f0, bnd, dp, x, h);
        h1[globalThreadIdx       ] = h[0];
        h1[globalThreadIdx + n1*4] = h[1];
        h1[globalThreadIdx + n1*8] = h[2];
        h1[globalThreadIdx + n1  ] = h1[globalThreadIdx + n1*3] = h[3];
        h1[globalThreadIdx + n1*2] = h1[globalThreadIdx + n1*6] = h[4];
        h1[globalThreadIdx + n1*5] = h1[globalThreadIdx + n1*7] = h[5];
    }
    else
    {
        h1[globalThreadIdx       ] = h1[globalThreadIdx + n1  ] = h1[globalThreadIdx + n1*2] = 
        h1[globalThreadIdx + n1*3] = h1[globalThreadIdx + n1*4] = h1[globalThreadIdx + n1*5] = 
        h1[globalThreadIdx + n1*6] = h1[globalThreadIdx + n1*7] = h1[globalThreadIdx + n1*8] = HIP_NAN_F;
    }
}


__global__ void push_element(float *f0, const float *phi, const float *f1)
{
    USIZE_t const globalThreadIdx = calculateGlobalIndex();
    float x[3];
    float fo;

    if(globalThreadIdx >= n1) return;
    fo   = f1[globalThreadIdx];               if(!isfinite(fo))   return;
    x[0] = phi[globalThreadIdx]       - 1.0f; if(!isfinite(x[0])) return;
    x[1] = phi[globalThreadIdx+n1]    - 1.0f; if(!isfinite(x[1])) return;
    x[2] = phi[globalThreadIdx+n1*2]  - 1.0f; if(!isfinite(x[2])) return;

    if(ext || (x[0]>=-0.01f && x[0]<=d0[0]-0.99f &&
               x[1]>=-0.01f && x[1]<=d0[1]-0.99f &&
               x[2]>=-0.01f && x[2]<=d0[2]-0.99f))
        push1(d0, f0, (float *)0, bnd, dp, x, fo);
}


__global__ void pushc_element(float *f0, float *c0, const float *phi, const float *f1)
{
    USIZE_t const globalThreadIdx = calculateGlobalIndex();
    float x[3];
    float fo;

    if(globalThreadIdx >= n1) return;
    fo   = f1[globalThreadIdx];              if(!isfinite(fo))   return;
    x[0] = phi[globalThreadIdx]      - 1.0f; if(!isfinite(x[0])) return;
    x[1] = phi[globalThreadIdx+n1]   - 1.0f; if(!isfinite(x[1])) return;
    x[2] = phi[globalThreadIdx+n1*2] - 1.0f; if(!isfinite(x[2])) return;

    if(ext || (x[0]>=-0.01f && x[0]<=d0[0]-0.99f &&
               x[1]>=-0.01f && x[1]<=d0[1]-0.99f &&
               x[2]>=-0.01f && x[2]<=d0[2]-0.99f))
        push1(d0, f0, c0, bnd, dp, x, fo);
}


__global__ void pushg_element(float *f0, const float *phi, const float *g1)
{
    USIZE_t const globalThreadIdx = calculateGlobalIndex();
    float x[3];
    float g[3];

    if(globalThreadIdx >= n1) return;
    x[0] = phi[globalThreadIdx]      - 1.0f;
    x[1] = phi[globalThreadIdx+n1]   - 1.0f;
    x[2] = phi[globalThreadIdx+n1*2] - 1.0f;

    if(VOXOK(x,d0))
    {
        g[0] = g1[globalThreadIdx];
        g[1] = g1[globalThreadIdx + n1];
        g[2] = g1[globalThreadIdx + n1*2];
        pushg1(d0, f0, bnd, dp, x, g);
    }
}


/* UNUSED */
__global__ void pushg3_element(float *g0, const float *phi, const float *f1)
{
    USIZE_t const globalThreadIdx = calculateGlobalIndex();
    float x[3];
    float f;

    if(globalThreadIdx >= n1) return;
    x[0] = phi[globalThreadIdx]      - 1.0f; if(!isfinite(x[0])) return;
    x[1] = phi[globalThreadIdx+n1]   - 1.0f; if(!isfinite(x[1])) return;
    x[2] = phi[globalThreadIdx+n1*2] - 1.0f; if(!isfinite(x[2])) return;

    if(ext || (x[0]>=-0.01f && x[0]<=d0[0]-0.99f &&
               x[1]>=-0.01f && x[1]<=d0[1]-0.99f &&
               x[2]>=-0.01f && x[2]<=d0[2]-0.99f))
    {
        f = f1[globalThreadIdx];  if(!ISFINITE(f)) return;
        pushg1a(d0, g0, bnd, dp, x, f);
    }
}



__global__ void affine_pull_element(float *f1, const float *f0)
{
    USIZE_t const globalThreadIdx = calculateGlobalIndex();
    int i,j,k,jk; 
    float x[3];

    i   = globalThreadIdx % d1[0];
    jk  = globalThreadIdx / d1[0];
    j   = jk % d1[1];
    k   = jk / d1[1]; if(k>=d1[2]) return;

    /* Assume Aff is adjusted for 0-offset data */
    x[0] = Aff[0]*i + Aff[3]*j + Aff[6]*k + Aff[ 9];
    x[1] = Aff[1]*i + Aff[4]*j + Aff[7]*k + Aff[10];
    x[2] = Aff[2]*i + Aff[5]*j + Aff[8]*k + Aff[11];

    if(VOXOK(x,d0))
        f1[globalThreadIdx] = pull1(d0, f0, bnd, dp, x);
    else
        f1[globalThreadIdx] = HIP_NAN_F;
}


/* WORK IN PROGRESS */
__global__ void affine_push_element(float *f0, const float *f1)
{
    USIZE_t const globalThreadIdx = calculateGlobalIndex();
    int i,j,k, jk;
    float x[3];
    float fo;

    i   = globalThreadIdx % d1[0];
    jk  = globalThreadIdx / d1[0];
    j   = jk % d1[1];
    k   = jk / d1[1]; if(k>=d1[2]) return;

    fo   = f1[globalThreadIdx]; if(!isfinite(fo)) return;

    /* Assume Aff is adjusted for 0-offset data */
    x[0] = Aff[0]*i + Aff[3]*j + Aff[6]*k + Aff[ 9];
    x[1] = Aff[1]*i + Aff[4]*j + Aff[7]*k + Aff[10];
    x[2] = Aff[2]*i + Aff[5]*j + Aff[8]*k + Aff[11];

    if(ISFINITE(x[0]) && ISFINITE(x[1]) && ISFINITE(x[2]) &&
       (ext || (x[0]>=-0.01f && x[0]<=d0[0]-0.99f &&
                x[1]>=-0.01f && x[1]<=d0[1]-0.99f &&
                x[2]>=-0.01f && x[2]<=d0[2]-0.99f)))
        push1(d0, f0, (float *)0, bnd, dp, x, fo);
}



