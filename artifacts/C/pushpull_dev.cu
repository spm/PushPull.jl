#define MAXD 5
#include "cuheader.h"
#include "bspline.cu"
#include "patch.cu"


__device__ float dotp_patch(const USIZE_t dp[], const float fp[], const float wi[], const float wj[], const float wk[])
{
    USIZE_t i, j, k;
    float fo = 0.0f;
    for(k=0; k<dp[2]; k++)
    {
        float wkk = wk[k];
        USIZE_t ok = dp[1]*k;
        for(j=0; j<dp[1]; j++)
        {
            float wjj = wkk*wj[j];
            USIZE_t oj = dp[0]*(j+ok);
            for(i=0; i<dp[0]; i++)
            {
                fo += wi[i]*wjj*fp[oj+i];
            }
        }
    }
    return fo;
}


/*  Does the equivalent of:
    g[0] = dotp_patch(dp, fp, dwi,  wj,  wk);
    g[1] = dotp_patch(dp, fp,  wi, dwj,  wk);
    g[2] = dotp_patch(dp, fp,  wi,  wj, dwk); */
__device__ void grad_patch(const USIZE_t dp[], const float fp[],
                           const float   wi[], const float  wj[], const float  wk[],
                           const float  dwi[], const float dwj[], const float dwk[], /*@out@*/float g[])
{
    USIZE_t i, j, k;
    g[0] = g[1] = g[2] = 0.0f;

    for(k=0; k<dp[2]; k++)
    {
        float wkk = wk[k], wkkz = dwk[k];
        USIZE_t ok = dp[1]*k;
        for(j=0; j<dp[1]; j++)
        {
            float wjjx = wkk*wj[j], wjjy = wkk*dwj[j], wjjz = wkkz*wj[j];
            USIZE_t oj  = dp[0]*(j+ok);
            for(i=0; i<dp[0]; i++)
            {
                float tmp = fp[oj+i];
                g[0] += dwi[i]*wjjx*tmp;
                tmp  *= wi[i];
                g[1] += wjjy*tmp;
                g[2] += wjjz*tmp;
            }
        }
    }
}


__device__ float pull1(const USIZE_t d0[], const float f0[], const int bnd[], const USIZE_t dp[], const float x[])
{
    SSIZE_t o[3];
    float wi[MAXD], wj[MAXD], wk[MAXD],  fp[MAXD*MAXD*MAXD];
    o[0] = weights(dp[0], x[0], wi);
    o[1] = weights(dp[1], x[1], wj);
    o[2] = weights(dp[2], x[2], wk);
    get_patch(dp, fp, bnd, o, d0, f0);
    return dotp_patch(dp, fp, wi, wj, wk);
}


__device__ void pullg1(const USIZE_t d0[], const float f0[], const int bnd[], const USIZE_t dp[], const float x[], /*@out@*/float g[])
{
    SSIZE_t o[3];
    float  wi[MAXD],  wj[MAXD],  wk[MAXD];
    float dwi[MAXD], dwj[MAXD], dwk[MAXD];
    float fp[MAXD*MAXD*MAXD];
    o[0] = weights(dp[0], x[0],  wi);
    o[1] = weights(dp[1], x[1],  wj);
    o[2] = weights(dp[2], x[2],  wk);
    (void)dweights(dp[0], x[0], dwi);
    (void)dweights(dp[1], x[1], dwj);
    (void)dweights(dp[2], x[2], dwk);

    get_patch(dp, fp, bnd, o, d0, f0);
    grad_patch(dp, fp, wi, wj, wk, dwi, dwj, dwk, g);
}


__device__ void pullh1(const USIZE_t d0[], const float f0[], const int bnd[], const USIZE_t dp[], const float x[], /*@out@*/float h[])
{
    SSIZE_t o[3];
    float  wi[MAXD],  wj[MAXD],  wk[MAXD];
    float dwi[MAXD], dwj[MAXD], dwk[MAXD];
    float hwi[MAXD], hwj[MAXD], hwk[MAXD];
    float fp[MAXD*MAXD*MAXD];
    o[0] = weights(dp[0], x[0],  wi);
    o[1] = weights(dp[1], x[1],  wj);
    o[2] = weights(dp[2], x[2],  wk);
    (void)dweights(dp[0], x[0], dwi);
    (void)dweights(dp[1], x[1], dwj);
    (void)dweights(dp[2], x[2], dwk);
    (void)hweights(dp[0], x[0], hwi);
    (void)hweights(dp[1], x[1], hwj);
    (void)hweights(dp[2], x[2], hwk);

    get_patch(dp, fp, bnd, o, d0, f0);
    /* Slow. Could be speeded up, as in grad_patch */
    h[0] = dotp_patch(dp, fp, hwi,  wj,  wk);
    h[1] = dotp_patch(dp, fp,  wi, hwj,  wk);
    h[2] = dotp_patch(dp, fp,  wi,  wj, hwk);
    h[3] = dotp_patch(dp, fp, dwi, dwj,  wk);
    h[4] = dotp_patch(dp, fp, dwi,  wj, dwk);
    h[5] = dotp_patch(dp, fp,  wi, dwj, dwk);
}


__device__ void weight_patch(const USIZE_t dp[], /*@OUT@*/float fp[], const float wi[], const float wj[], const float wk[], const float fo)
{
    USIZE_t i, j, k;
    for(k=0; k<dp[2]; k++)
    {
        float wkk = wk[k];
        USIZE_t ok = dp[1]*k;
        for(j=0; j<dp[1]; j++)
        {
            float wjj = wkk*wj[j];
            USIZE_t oj = dp[0]*(j+ok);
            for(i=0; i<dp[0]; i++)
            {
                float wt = wi[i]*wjj;
                fp[oj+i] = wt*fo;
            }
        }
    }
}


__device__ void push1(const USIZE_t d0[], float f0[], /*@NULL@*/float c0[], const int bnd[], const USIZE_t dp[], const float x[], const float fo)
{
    SSIZE_t o[3];
    float wi[MAXD], wj[MAXD], wk[MAXD], fp[MAXD*MAXD*MAXD];
    o[0] = weights(dp[0], x[0], wi);
    o[1] = weights(dp[1], x[1], wj);
    o[2] = weights(dp[2], x[2], wk);
    weight_patch(dp, fp, wi, wj, wk, fo);
    put_patch(dp, fp, bnd, o, d0, f0);
    if(c0!=(void *)0)
    {
        weight_patch(dp, fp, wi, wj, wk, 1.0f);
        put_patch(dp, fp, bnd, o, d0, c0);
    }
}


__device__ void pushg1(const USIZE_t d0[], float f0[], const int bnd[], const USIZE_t dp[], const float x[], const float g[])
{
    SSIZE_t o[3];
    float  wi[MAXD],  wj[MAXD],  wk[MAXD];
    float dwi[MAXD], dwj[MAXD], dwk[MAXD];
    float fp[MAXD*MAXD*MAXD];
    o[0] = weights(dp[0], x[0],  wi);
    o[1] = weights(dp[1], x[1],  wj);
    o[2] = weights(dp[2], x[2],  wk);
    (void)dweights(dp[0], x[0], dwi);
    (void)dweights(dp[1], x[1], dwj);
    (void)dweights(dp[2], x[2], dwk);

    /* Slightly inefficient */
    weight_patch(dp, fp, dwi, wj, wk, g[0]);
    put_patch(dp, fp, bnd, o, d0, f0);

    weight_patch(dp, fp, wi, dwj, wk, g[1]);
    put_patch(dp, fp, bnd, o, d0, f0);

    weight_patch(dp, fp, wi, wj, dwk, g[2]);
    put_patch(dp, fp, bnd, o, d0, f0);
}


/* UNUSED */
__device__ void pushg1a(const USIZE_t d0[], float g0[], const int bnd[], const USIZE_t dp[], const float x[], const float f)
{
    SSIZE_t o[3], n0 = d0[0]*d0[1]*d0[2];
    float  wi[MAXD],  wj[MAXD],  wk[MAXD];
    float dwi[MAXD], dwj[MAXD], dwk[MAXD];
    float fp[MAXD*MAXD*MAXD];
    o[0] = weights(dp[0], x[0],  wi);
    o[1] = weights(dp[1], x[1],  wj);
    o[2] = weights(dp[2], x[2],  wk);
    (void)dweights(dp[0], x[0], dwi);
    (void)dweights(dp[1], x[1], dwj);
    (void)dweights(dp[2], x[2], dwk);

    /* Slightly inefficient */
    weight_patch(dp, fp, dwi, wj, wk, f);
    put_patch(dp, fp, bnd, o, d0, g0);

    weight_patch(dp, fp, wi, dwj, wk, f);
    put_patch(dp, fp, bnd, o, d0, g0+n0);

    weight_patch(dp, fp, wi, wj, dwk, f);
    put_patch(dp, fp, bnd, o, d0, g0+n0*2);
}


/* WORK IN PROGRESS */
__device__ void shootfun1_dev(const USIZE_t i, const USIZE_t j, const USIZE_t k, const USIZE_t d[], float u1[], const float u0[], const float v[], const int bnd[], const float s)
{
    SSIZE_t o[3];
    USIZE_t dp[3], n0 = d[0]*d[1]*d[2], oo = i+d[0]*(j+d[1]*k);
    float   x[3], uc[3], g[3],
	    J[3*3];    /* Gradients go into Jacobain matrix */

    float   s2 = -s*0.5f; /* account for two voxel spacing when computing gradients */

    o[0]     = i;
    o[1]     = j;
    o[2]     = k;

    x[0]     = diffs(g, bnd  , o, d, v     )*s + i;
    J[0+3*0] = s2*g[0] + 1.0f;
    J[0+3*1] = s2*g[1];
    J[0+3*2] = s2*g[2];

    x[1]     = diffs(g, bnd+3, o, d, v+n0  )*s + j;
    J[1+3*0] = s2*g[0];
    J[1+3*1] = s2*g[1] + 1.0f;
    J[1+3*2] = s2*g[2];

    x[2]     = diffs(g, bnd+6, o, d, v+n0*2)*s + k;
    J[2+3*0] = s2*g[0];
    J[2+3*1] = s2*g[1];
    J[2+3*2] = s2*g[2] + 1.0f;


    /* Get element of u0 at this point */
    uc[0] = u0[oo];
    uc[1] = u0[oo+n0];
    uc[2] = u0[oo+n0*2];

    /* Transform uc by Jacobain and push into u1 at position x (slow because of atomic additions) */
    dp[0] = dp[1] = dp[2] = 2; /* Trilinear */
    push1(d, u1     , (float *)NULL, bnd  , dp, x, J[0+3*0]*uc[0] + J[1+3*0]*uc[1] + J[2+3*0]*uc[2]);
    push1(d, u1+n0  , (float *)NULL, bnd+3, dp, x, J[0+3*1]*uc[0] + J[1+3*1]*uc[1] + J[2+3*1]*uc[2]);
    push1(d, u1+n0*2, (float *)NULL, bnd+6, dp, x, J[0+3*2]*uc[0] + J[1+3*2]*uc[1] + J[2+3*2]*uc[2]);
}


/* WORK IN PROGRESS */
__device__ void comp1_dev(const USIZE_t d[], float ff[], const float f0[], const int bnd[], const float x[])
{
    SSIZE_t o[3];
    USIZE_t n = d[0]*d[1]*d[2], dp[3];
    float wi[2], wj[2], wk[2],  fp[2*2*2];
    dp[0] = dp[1] = dp[2] = 2;
    o[0]  = weights(dp[0], x[0], wi);
    o[1]  = weights(dp[1], x[1], wj);
    o[2]  = weights(dp[2], x[2], wk);

    get_patch(dp, fp, bnd+3*0, o, d, f0    ); ff[0] = dotp_patch(dp, fp, wi, wj, wk);
    get_patch(dp, fp, bnd+3*1, o, d, f0+n  ); ff[1] = dotp_patch(dp, fp, wi, wj, wk);
    get_patch(dp, fp, bnd+3*2, o, d, f0+n*2); ff[2] = dotp_patch(dp, fp, wi, wj, wk);
}


