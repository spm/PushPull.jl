#include "hip/hip_runtime.h"
#define CUDA
#include "cuheader.h"
#include "pushpull_dev.cu"
/* #include<hip/hip_math_constants.h> */

/* Use constant memory for lower access times */
__constant__ int     bnd[3*3]; /* boundary codes   */
__constant__ USIZE_t d[3];     /* image dimensions */
__constant__ float   s;        /* scaling of velocities */

__global__ void shootfun1_element(float *u1, const float *u0, const float *v0)
{
    USIZE_t i,j,k;

    i = (blockIdx.x*blockDim.x + threadIdx.x); if (i>=d[0]) return;
    j = (blockIdx.y*blockDim.y + threadIdx.y); if (j>=d[1]) return;
    k = (blockIdx.z*blockDim.z + threadIdx.z); if (k>=d[2]) return;

    shootfun1_dev(i, j, k, d, u1, u0, v0, bnd, s);
}

__global__ void shootfun2_element(float *psi1, const float *psi0, const float *v)
{
    USIZE_t i,j,k,o,n;
    float x[3], ff[3];

    i = (blockIdx.x*blockDim.x + threadIdx.x); if (i>=d[0]) return;
    j = (blockIdx.y*blockDim.y + threadIdx.y); if (j>=d[1]) return;
    k = (blockIdx.z*blockDim.z + threadIdx.z); if (k>=d[2]) return;
    o = i+d[0]*(j+d[1]*k);
    n = d[0]*d[1]*d[2];

    x[0] = (float)i-v[o+n*0]*s;
    x[1] = (float)j-v[o+n*1]*s;
    x[2] = (float)k-v[o+n*2]*s;
    comp1_dev(d, ff, psi0, bnd, x);
    psi1[o+n*0] = ff[0]-v[o+n*0]*s;
    psi1[o+n*1] = ff[1]-v[o+n*1]*s;
    psi1[o+n*2] = ff[2]-v[o+n*2]*s;
}

